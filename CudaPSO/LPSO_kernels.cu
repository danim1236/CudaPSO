#include "hip/hip_runtime.h"
#include "LPSO_kernels.cuh"

#include <algorithm>


extern "C" __device__ __device_builtin__ void __syncthreads();
extern "C" __device__ __device_builtin__ float fminf(float x, float y);
extern "C" __device__ __device_builtin__ float fmaxf(float x, float y);
extern "C" __device__ __device_builtin__ unsigned int __uAtomicInc(unsigned int *address, unsigned int val);
extern "C" __device__ __device_builtin__ void __threadfence_system(void);


template<class T>
__global__ void k_LPSOInit(
	int numParticles, 
	int numDimensions, 
	float *_positions, 
	float *_velocities, 
	float *_bestPositions,
	float *_bestFitness,
	hiprandState *_s)
{
	__shared__ float bestFitness[1024];
	__shared__ T Eval;

	int idx = threadIdx.x;
	int ptr_g0 = blockDim.x * blockIdx.x;
	int gidx = ptr_g0 + idx;
	if (gidx >= numParticles)
		bestFitness[idx] = FLT_MAX;
	__syncthreads();

	int ptr_g = gidx * numDimensions; // posi��o na memoria
	if (gidx < numParticles)
	{
		hiprand_init(threadIdx.x, 0, 0, &_s[idx]);

		// Calculate randon pos & vel
		for (int d = 0; d < numDimensions; ++d)
		{
			float min = _c_LPSOMinPosition[d];
			float max = _c_LPSOMaxPosition[d];
			_positions[ptr_g + d] = hiprand_uniform(&_s[idx])*(max - min) + min;
			_velocities[ptr_g + d] = hiprand_uniform(&_s[idx])*(max - min) + min;
		}

		// Initizalizes local bests
		bestFitness[idx] = Eval(&_positions[ptr_g]);
	}
	__syncthreads();

	if (gidx < numParticles)
	{
		for (int d = 0; d < numDimensions; ++d)
			_bestPositions[ptr_g + d] = _positions[ptr_g + d];
		_bestFitness[gidx] = bestFitness[idx];
	}
}

template<class T>
__global__ void k_LPSOIterateMultiBlock(
	int numParticles, 
	int numDimensions, 
	float *_positions, 
	float *_velocities, 
	float *_bestPositions,
	float *_bestFitness,
	hiprandState *_s)
{
	__shared__ T Eval;

	int p = threadIdx.x;
	int block = blockIdx.x;
	int ptr_g0 = blockDim.x * block;
	int gp = ptr_g0 + p;
	int lastP = fminf(numParticles - ptr_g0, 1024);

	__syncthreads();

	int ptr_g = gp * numDimensions;
	if (gp < numParticles)
	{
		int prev = (p + numParticles - 1) % numParticles;
		int next = (p + 1) % numParticles;
		int best = gp;
		if (_bestFitness[prev] < _bestFitness[best])
			best = prev;
		if (_bestFitness[next] < _bestFitness[best])
			best = prev;

		for (int d = 0; d < numDimensions; ++d)
		{
			float r1 = hiprand_uniform(&_s[p]);
			float r2 = hiprand_uniform(&_s[p]);

			int ptr = ptr_g + d;
			float position = _positions[ptr];
			float newVelocity = (W * _velocities[ptr]) +
				(C1 * r1 * (_bestPositions[ptr] - position)) +
				(C2 * r2 * (_bestPositions[best * numDimensions + d] - position));

			newVelocity = fmaxf(_c_LPSOMinPosition[d], fminf(_c_LPSOMaxPosition[d], newVelocity));
			_velocities[ptr] = newVelocity;

			float newPosition = position + newVelocity;
			newPosition = fmaxf(_c_LPSOMinPosition[d], fminf(_c_LPSOMaxPosition[d], newPosition));
			_positions[ptr] = newPosition;
		}
		float newFitness = Eval(&_positions[ptr_g]);
		if (newFitness < _bestFitness[gp])
		{
			_bestFitness[gp] = newFitness;
			for (int d = 0; d < numDimensions; ++d)
			{
				int ptr = ptr_g + d;
				_bestPositions[ptr] = _positions[ptr];
			}
		}
	}
	__syncthreads();
}


__global__ void k_LPSOInterMinimum(
	int numParticles, 
	int numDimensions, 
	float *_bestPositions,
	float *_bestFitness,
	float *_bestGlobalPosition,
	float *_bestGlobalFitness,
	hiprandState *_s)
{
	__shared__ int ptrs[1024];
	__shared__ float bestFitness[1024];

	int p = threadIdx.x;
	int block = blockIdx.x;
	int ptr_g0 = blockDim.x * block;
	int gp = ptr_g0 + p;
	if (gp < numParticles)
		bestFitness[p] = _bestFitness[gp];
	else
		bestFitness[p] = FLT_MAX;
	__syncthreads();

	ptrs[p] = p;
	__syncthreads();
	for (int s = blockDim.x / 2; s > 0; s /= 2)
	{
		if (p < s)
		{
			if (bestFitness[ptrs[p]] > bestFitness[ptrs[p + s]])
			{
				int tmp = ptrs[p + s];
				ptrs[p + s] = ptrs[p];
				ptrs[p] = tmp;
			}
		}
		__syncthreads();
	}
	
	if (p < numDimensions)
		_bestGlobalPosition[block * numDimensions + p] = _bestPositions[(ptr_g0 + ptrs[0]) * numDimensions + p];
	if (p == 0)
		_bestGlobalFitness[block] = bestFitness[ptrs[0]];
}


__global__ void k_LPSOMinimum(int _numBlocks, int numDimensions, float *_position, float *_fitness)
{
	__shared__ float fitness[1024];
	__shared__ int ptrs[1024];

	int idx = threadIdx.x;
	ptrs[idx] = idx;
	if (idx >= _numBlocks)
		fitness[idx] = FLT_MAX;
	__syncthreads();

	if (idx < _numBlocks)
		fitness[idx] = _fitness[idx];
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s /= 2)
	{
		if (idx < s)
		{
			if (fitness[ptrs[idx]] > fitness[ptrs[idx + s]])
			{
				int tmp = ptrs[idx + s];
				ptrs[idx + s] = ptrs[idx];
				ptrs[idx] = tmp;
			}
		}
		__syncthreads();
	}
	if (idx < numDimensions)
		_position[idx] = _position[ptrs[0] * numDimensions + idx];
	if (idx == 0)
		_fitness[0] = _fitness[ptrs[0]];
}
