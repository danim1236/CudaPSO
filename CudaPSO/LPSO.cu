#include "hip/hip_runtime.h"
#include "LPSO.cuh"

#include "LPSO_kernels.cu"
#include "Functors.cuh"

template<typename T>
LPSO<T>::LPSO(int numParticles, int numIterations)
:
PSOBase(numParticles, numIterations, T::GetNumDimensions(), T::GetMinPosition(), T::GetMaxPosition()),
_d_positions(_positions.size()),
_d_velocities(_velocities.size()),
_d_minPositions(_minPositions),
_d_maxPositions(_maxPositions),
_d_bestPositions(_bestPositions.size()),
_d_bestFitness(_bestFitness.size()),
_d_state(numParticles)
{
	if (_numDimensions > MAX_DIMENSIONS)
		throw new exception("_numDimensions > MAX_DIMENSIONS");
	CalculateGeometry();
	_d_bestGlobalPosition.resize(_numDimensions * _numBlocks);
	_d_bestGlobalFitness.resize(_numBlocks);
	_bestGlobalPosition.resize(_numDimensions * _numBlocks);
	_bestGlobalFitness.resize(_numBlocks);
	hipMemcpyToSymbol(HIP_SYMBOL(_c_LPSOMinPosition), _minPositions.data(), _minPositions.size() * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(_c_LPSOMaxPosition), _maxPositions.data(), _maxPositions.size() * sizeof(float));
}

template<typename T>
void LPSO<T>::Init()
{
	int threadNumber = pow(2, ceil(log(_numThreads)/log(2)));
	int blockNumber = pow(2, ceil(log(_numBlocks)/log(2)));
	k_LPSOInit<T><<<_numBlocks, threadNumber>>>(_numParticles, _numDimensions,
		raw_pointer_cast(_d_positions.data()), 
		raw_pointer_cast(_d_velocities.data()), 
		raw_pointer_cast(_d_bestPositions.data()),
		raw_pointer_cast(_d_bestFitness.data()),
		raw_pointer_cast(_d_state.data()));
	hipDeviceSynchronize();
	k_LPSOMinimum<<<1, blockNumber>>>(_numBlocks, _numDimensions,
		raw_pointer_cast(_d_bestGlobalPosition.data()),
		raw_pointer_cast(_d_bestGlobalFitness.data()));
	UpdateHost();
}

template<typename T>
void LPSO<T>::Iterate()
{
	int threadNumber = pow(2, ceil(log(_numThreads)/log(2)));
	int blockNumber = pow(2, ceil(log(_numBlocks)/log(2)));
	for (int i = 0; i < _numIterations; ++i)
	{
		k_LPSOIterateMultiBlock<T><<<_numBlocks, threadNumber>>>(_numParticles, _numDimensions,
			raw_pointer_cast(_d_positions.data()), 
			raw_pointer_cast(_d_velocities.data()), 
			raw_pointer_cast(_d_bestPositions.data()),
			raw_pointer_cast(_d_bestFitness.data()),
			raw_pointer_cast(_d_state.data()));
	}
	k_LPSOInterMinimum<<<_numBlocks, threadNumber>>>(_numParticles, _numDimensions,
		raw_pointer_cast(_d_bestPositions.data()),
		raw_pointer_cast(_d_bestFitness.data()),
		raw_pointer_cast(_d_bestGlobalPosition.data()),
		raw_pointer_cast(_d_bestGlobalFitness.data()),
		raw_pointer_cast(_d_state.data()));
	if (blockNumber > 1)
		k_LPSOMinimum<<<1, blockNumber>>>(_numBlocks, _numDimensions,
			raw_pointer_cast(_d_bestGlobalPosition.data()),
			raw_pointer_cast(_d_bestGlobalFitness.data()));

	_iteration += _numIterations;
	UpdateHost();
}

template<typename T>
void LPSO<T>::UpdateHost()
{
	_positions = _d_positions;
	_velocities = _d_velocities;
	_minPositions = _d_minPositions;
	_maxPositions = _d_maxPositions;
	_bestPositions = _d_bestPositions;
	_bestFitness = _d_bestFitness;
	_bestGlobalPosition = _d_bestGlobalPosition;
	_bestGlobalFitness = _d_bestGlobalFitness;
}

template<typename T>
void LPSO<T>::CalculateGeometry()
{
	int numDevices;
	hipGetDeviceCount(&numDevices);
	if (numDevices < 1)
		throw std::exception("Nenhum dispositivo cuda");

	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);

	int maxThreads = devProp.maxThreadsPerBlock;

	_numThreads = (_numParticles + 31 ) / 32 * 32;
	_numThreads = std::min(((_numThreads + 31)/32)*32, maxThreads);
	_numBlocks = (_numParticles + _numThreads - 1) / _numThreads;
}
